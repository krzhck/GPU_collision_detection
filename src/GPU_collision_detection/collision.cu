#include "hip/hip_runtime.h"
#include"collision.cuh"
#include "ball.hpp"


//通用函数

__device__ float Dist(float x, float y, float z)
{
	return sqrt(x * x + y * y + z * z);
}

__device__ float Dist(Coord & p)
{
	return sqrt(p.x * p.x + p.y * p.y + p.z * p.z);
}

__device__ float Multiply(Coord & a, Coord& b)
{
	return (a.x * b.x + a.y * b.y + a.z * b.z);
}

/*
	描述：�?�理与边界相�?
	参数：X范围�?-X, X), Z范围(-Z, Z), Y范围(0, Y)
	返回：无
*/
__device__ void HandleCollisionWall(Ball& ball, float XRange, float ZRange, float Height)
{
	if (ball.Pos.x - ball.Radius < -XRange)
	{
		ball.Pos.x = -XRange + ball.Radius;
		ball.Speed.x = -ball.Speed.x;
	}
	else if (ball.Pos.x + ball.Radius > XRange)
	{
		ball.Pos.x = XRange - ball.Radius;
		ball.Speed.x = -ball.Speed.x;
	}
	if (ball.Pos.z - ball.Radius < -ZRange)
	{
		ball.Pos.z = -ZRange + ball.Radius;
		ball.Speed.z = -ball.Speed.z;
	}
	else if (ball.Pos.z + ball.Radius > ZRange)
	{
		ball.Pos.z = ZRange - ball.Radius;
		ball.Speed.z = -ball.Speed.z;
	}
	if (ball.Pos.y - ball.Radius < 0)
	{
		ball.Pos.y = ball.Radius;
		ball.Speed.y = -ball.Speed.y;
	}
	else if (ball.Pos.y + ball.Radius > Height)
	{
		ball.Pos.y = Height - ball.Radius;
		ball.Speed.y = -ball.Speed.y;
	}
}


/*
	描述：�?�理小球�?行运动和与边界�?�撞
	参数：单次运动时间，X范围�?-X, X), Z范围(-Z, Z), Y范围(0, Y)
	返回：无
*/
__device__ void BallMove(Ball& ball, float time, float XRange, float ZRange, float Height)
{

	ball.Pos.x = ball.Pos.x + ball.Speed.x * time;
	ball.Pos.y = ball.Pos.y + ball.Speed.y * time;
	ball.Pos.z = ball.Pos.z + ball.Speed.z * time;
	HandleCollisionWall(ball, XRange, ZRange, Height);
}

/*
	描述：判�?两个球是否相�?
	参数：球a，球b
	返回：是1，否0
*/
__device__ bool JudgeCollision(Ball& a, Ball& b)
{
	float dist = 0;
	float dist_x = a.Pos.x - b.Pos.x;
	float dist_y = a.Pos.y - b.Pos.y;
	float dist_z = a.Pos.z - b.Pos.z;
	dist = Dist(dist_x, dist_y, dist_z);
	if (dist < a.Radius + b.Radius)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

/*
	描述：两球相撞后更新速度
	参数：球a，球b
	返回：无
*/
__device__ void ChangeSpeed(Ball& a, Ball& b)
{
	//径向速度按照质量做变�?，法向速度不变
	float dist = 0;
	float diff_x = b.Pos.x - a.Pos.x;
	float diff_y = b.Pos.y - a.Pos.y;
	float diff_z = b.Pos.z - a.Pos.z;
	dist = Dist(diff_x, diff_y, diff_z);

	//求径向，法向速度
	float rate_collide_a = (a.Speed.x * diff_x + a.Speed.y * diff_y + a.Speed.z * diff_z) / dist / dist;
	float speed_collide_a_x = diff_x * rate_collide_a;
	float speed_collide_a_y = diff_y * rate_collide_a;
	float speed_collide_a_z = diff_z * rate_collide_a;

	float rate_collide_b = (b.Speed.x * diff_x + b.Speed.y * diff_y + b.Speed.z * diff_z) / dist / dist;
	float speed_collide_b_x = diff_x * rate_collide_b;
	float speed_collide_b_y = diff_y * rate_collide_b;
	float speed_collide_b_z = diff_z * rate_collide_b;

	float unchanged_a_x = a.Speed.x - speed_collide_a_x;
	float unchanged_a_y = a.Speed.y - speed_collide_a_y;
	float unchanged_a_z = a.Speed.z - speed_collide_a_z;

	float unchanged_b_x = b.Speed.x - speed_collide_b_x;
	float unchanged_b_y = b.Speed.y - speed_collide_b_y;
	float unchanged_b_z = b.Speed.z - speed_collide_b_z;


	//假�?�b不动，a撞b，更新两者径向速度
	float speed_collide_new_a_x = (speed_collide_a_x * (a.Weight - b.Weight) + speed_collide_b_x * (2 * b.Weight)) / (a.Weight + b.Weight);
	float speed_collide_new_a_y = (speed_collide_a_y * (a.Weight - b.Weight) + speed_collide_b_y * (2 * b.Weight)) / (a.Weight + b.Weight);
	float speed_collide_new_a_z = (speed_collide_a_z * (a.Weight - b.Weight) + speed_collide_b_z * (2 * b.Weight)) / (a.Weight + b.Weight);

	float speed_collide_new_b_x = (speed_collide_a_x * (2 * a.Weight) + speed_collide_b_x * (b.Weight - a.Weight)) / (a.Weight + b.Weight);
	float speed_collide_new_b_y = (speed_collide_a_y * (2 * a.Weight) + speed_collide_b_y * (b.Weight - a.Weight)) / (a.Weight + b.Weight);
	float speed_collide_new_b_z = (speed_collide_a_z * (2 * a.Weight) + speed_collide_b_z * (b.Weight - a.Weight)) / (a.Weight + b.Weight);

	a.Speed.x = speed_collide_new_a_x + unchanged_a_x;
	a.Speed.y = speed_collide_new_a_y + unchanged_a_y;
	a.Speed.z = speed_collide_new_a_z + unchanged_a_z;

	b.Speed.x = speed_collide_new_b_x + unchanged_b_x;
	b.Speed.y = speed_collide_new_b_y + unchanged_b_y;
	b.Speed.z = speed_collide_new_b_z + unchanged_b_z;
}

/*
描述：在球之间的碰撞检测完成后，�?�理球的运动以及和边界的碰撞（并行）
参数：球列表，一次的时间，X范围(-X,X),Z范围(-Z,Z),Y范围(0,Y)，球�?�?
返回：无，但�?更新球列�?
*/
__global__ void UpdateBallsMove(Ball* balls, float TimeOnce, float XRange, float ZRange, float Height, int N)
{
	// 获取全局索引
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// 步长
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += stride)
	{
		BallMove(balls[i], TimeOnce, XRange, ZRange, Height);
	}

}


//空间划分算法相关函数
/*
描述：初始化cells，objects数组，前者�?�录物体所在的格子信息（格子x，y，z的id，home还是phantom），后者�?�录物体id和home/phantom
参数：空的cell，phantom；球列表和个数，还有各�?�格子信�?
返回：更新cells，objects数组和cell_num
*/
__global__ void InitCellKernel(uint32_t *cells, uint32_t *objects, Ball* balls, int N, float XRange, float ZRange, float Height, float GridSize, int GridX, int GridY, int GridZ) 
{
	unsigned int count = 0;

	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += gridDim.x * blockDim.x)
	{
		int current_cell_id = i * 8; //每个球最多在8�?格子�?
		int cell_info = 0;
		int object_info = 0;
		int current_count = 0;
		float x = balls[i].Pos.x;
		float y = balls[i].Pos.y;
		float z = balls[i].Pos.z;
		float radius = balls[i].Radius;

		//找到home cell
		int hash_x = (x + XRange) / GridSize;
		int hash_y = (y) / GridSize;
		int hash_z = (z + ZRange) / GridSize;
		cell_info = hash_x << 17 | hash_y << 9 | hash_z << 1 | HOME_CELL;
		object_info = i << 1 | HOME_OBJECT;
		cells[current_cell_id] = cell_info;
		objects[current_cell_id] = object_info;
		current_cell_id++;
		count++;
		current_count++;

		//找phantom
		for (int dx = -1; dx <= 1; dx++)
		{
			for (int dy = -1; dy <= 1; dy++)
			{
				for (int dz = -1; dz <= 1; dz++)
				{
					int new_hash_x = hash_x + dx;
					int new_hash_y = hash_y + dy;
					int new_hash_z = hash_z + dz;

					//�?己不考虑
					if (dx == 0 && dy == 0 && dz == 0)
					{
						continue;
					}

					//越界不考虑
					if (new_hash_x < 0 || new_hash_x >= GridX ||
						new_hash_y < 0 || new_hash_y >= GridY ||
						new_hash_z < 0 || new_hash_z >= GridZ)
					{
						continue;
					}

					float relative_x = 0;
					float relative_y = 0;
					float relative_z = 0;
					if (dx == 0)
					{
						relative_x = x;
					}
					else if (dx == -1)
					{
						relative_x = hash_x * GridSize - XRange;
					}
					else
					{
						relative_x = (hash_x + 1) * GridSize - XRange;
					}

					if (dz == 0)
					{
						relative_z = z;
					}
					else if (dz == -1)
					{
						relative_z = hash_z * GridSize - ZRange;
					}
					else
					{
						relative_z = (hash_z + 1) * GridSize - ZRange;
					}

					if (dy == 0)
					{
						relative_y = y;
					}
					else if (dy == -1)
					{
						relative_y = hash_y * GridSize;
					}
					else
					{
						relative_y = (hash_y + 1) * GridSize;
					}

					relative_x -= x;
					relative_y -= y;
					relative_z -= z;

					float dist = Dist(relative_x, relative_y, relative_z);
					if (dist < radius)
					{
						int cell_info = new_hash_x << 17 | new_hash_y << 9 | new_hash_z << 1 | PHANTOM_CELL;
						int object_info = i << 1 | PHANTOM_OBJECT;
						cells[current_cell_id] = cell_info;
						objects[current_cell_id] = object_info;
						current_cell_id++;
						count++;
						current_count++;
					}
				}
			}
		}

		//补齐
		while (current_count < 8)
		{

			cells[current_cell_id] = UINT32_MAX;
			objects[current_cell_id] = i << 2;
			current_cell_id++;
			current_count++;
		}

	}

}


/*
描述：�?�算前i和的算法
参数：原始数组，�?数n
返回：原始数组变成前i�?和数�?
*/
__device__ void PrefixSum(uint32_t *values, unsigned int n) 
{
	int offset = 1;
	int a;
	uint32_t temp;

	//reduction
	for (int d = n / 2; d; d /= 2) 
	{
		__syncthreads();

		if (threadIdx.x < d) 
		{
			a = (threadIdx.x * 2 + 1) * offset - 1;
			values[a + offset] += values[a];
		}

		offset *= 2;
	}

	if (!threadIdx.x) 
	{
		values[n - 1] = 0;
	}

	//reverse
	for (int d = 1; d < n; d *= 2) 
	{
		__syncthreads();
		offset /= 2;

		if (threadIdx.x < d) 
		{
			a = (threadIdx.x * 2 + 1) * offset - 1;
			temp = values[a];
			values[a] = values[a + offset];
			values[a + offset] += temp;
		}
	}
}

/*
描述：�?�cells求前缀�?
参数：cells，待更新前缀和，N个cell，偏移量
返回：更新前缀�?
*/
__global__ void GetRadixSum(uint32_t *cells, uint32_t *radix_sums, int N, int shift)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int num_indices = 1 << RADIX_LENGTH;


	//初�?�化
	for (int i = index; i < num_indices; i++)
	{
		radix_sums[i] = 0;
	}
	__syncthreads();


	 //求和
	for (int i = index; i < N; i += stride)
	{
		//非常重�?�，不这样做无法有效求和
		for (int j = 0; j < blockDim.x; j++)
		{
			if (threadIdx.x % blockDim.x == j)
			{
				int current_radix_num = (cells[i] >> shift) & (num_indices - 1);
				radix_sums[current_radix_num] ++;
			}
		}

	}
	__syncthreads();
	//求前缀�?
	PrefixSum(radix_sums, num_indices);
	__syncthreads();
}

/*
描述：重新分配元�?
参数：cells，object数组，他�?待更新的分配结果temp，前缀和数组，N�?元素，偏移量，每�?线程处理几个cell
返回：更新前缀�?
*/
__global__ void RearrangeCell(uint32_t *cells, uint32_t *objects, uint32_t *cells_temp, uint32_t *objects_temp, uint32_t *radix_sums, int N, int shift)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int num_radices = 1 << RADIX_LENGTH;

	if (index != 0) return;
	//分配
	for (int i = 0; i < N; i ++ )
	{
		int current_radix_num = (cells[i] >> shift) & (num_radices - 1);
		cells_temp[radix_sums[current_radix_num]] = cells[i];
		objects_temp[radix_sums[current_radix_num]] = objects[i];
		radix_sums[current_radix_num] ++;
	}
}

/*
描述：获取排序后数组的index（cell变化的位�?�?
参数：cell，cell�?数N,待更新的indice，待更新的indice�?�?
返回：无，但�?更新indice数组和indice�?�?
*/
__global__ void GetCellIndex(uint32_t *cells, int N, uint32_t* indices, uint32_t* num_indices)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	//�?能串�?
	if (index != 0) return;
	num_indices[0] = 0;
	uint32_t mask = (1 << 24) - 1;
	uint32_t previous = UINT32_MAX;
	uint32_t current = UINT32_MAX;
	for (int i = 0; i < N; i++)
	{
		current = mask & (cells[i] >> 1);
		if (previous == UINT32_MAX)
		{
			previous = current;
		}
		if (previous != current)
		{
			indices[num_indices[0]] = i;
			num_indices[0]++;
		}
		previous = current;
	}
	indices[num_indices[0]] = N;
	num_indices[0]++;
}


/*
描述：�?�cell，object做基数排序，并且获取index（cell变化的位�?�?
参数：cell，object数组；他�?的temp形式用于排序；待求的前缀和数组；cell�?数；待求的index数组和长度；线程情况
返回：无，但�?更新cell，object数组，还有index数组和其长度
*/
void SortCells(uint32_t *cells, uint32_t *objects, uint32_t *cells_temp, uint32_t *objects_temp,
	uint32_t *radix_sums, int N, uint32_t* indices, uint32_t* num_indices,
	unsigned int num_blocks, unsigned int threads_per_block)
{
	uint32_t *cells_swap;
	uint32_t *objects_swap;
	for (int i = 0; i < 32; i += RADIX_LENGTH)
	{
		//求前缀�?
		GetRadixSum <<< num_blocks, threads_per_block >>> (cells, radix_sums, N, i);

		//用前缀和重新分�?
		RearrangeCell <<< num_blocks, threads_per_block >>> (cells, objects, cells_temp, objects_temp,
			radix_sums, N, i);
		
		//交换原�?�和temp
		cells_swap = cells;
		cells = cells_temp;
		cells_temp = cells_swap;
		objects_swap = objects;
		objects = objects_temp;
		objects_temp = objects_swap;
	}
	GetCellIndex <<< num_blocks, threads_per_block >>> (cells, N, indices, num_indices);
}

/*
描述：cuda碰撞检测和处理函数
参数：cell和object数组，ball数组，球和cell的个数，index数组和个数，线程信息，场�?的各种限制和格子信息
返回：无，但�?进�?��?�撞检测和处理
*/
__global__ void HandleCollisionCuda(uint32_t *cells, uint32_t *objects, Ball* balls, int num_balls, int num_cells,
	uint32_t* indices, uint32_t num_indices, unsigned int group_per_thread,
	float XRange, float ZRange, float Height, float GridSize, int GridX, int GridY, int GridZ)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	for (int group_num = 0; group_num < group_per_thread; group_num++)
	{
		//判断�?否越界，找到处理的start，end
		int cell_id = index * group_per_thread + group_num;
		if (cell_id >= num_indices)
		{
			break;
		}
		int end = indices[cell_id];
		int start = 0;
		if (cell_id == 0)
		{
			start = 0;
		}
		else
		{
			start = indices[cell_id - 1];
		}

		//找其中home的个�?
		int home_num = 0;
		for (int i = start; i < end; i++)
		{
			int type = cells[i] & 1;
			if (type == HOME_CELL)
			{
				home_num++;
			}
			else
			{
				break;
			}
		}

		//遍历碰撞检�?
		for (int i = start; i < start + home_num; i++)
		{
			if (cells[i] == UINT32_MAX) break;
			int ball_i = (objects[i] >> 1) & 65535;
			
			for (int j = i + 1; j < end; j++)
			{
				if (cells[j] == UINT32_MAX) break;
				int ball_j = (objects[j] >> 1) & 65535;

				//2个home，直接�?�撞检�?
				if (j < start + home_num)
				{
					if (JudgeCollision(balls[ball_i], balls[ball_j]))
					{
						ChangeSpeed(balls[ball_i], balls[ball_j]);
					}
				}

				//home和phantom，需要判�?
				else
				{
					int home_i = (cells[i] >> 1) & ((1 << 24) - 1);
					int j_x = (balls[ball_j].Pos.x + XRange) / GridSize;
					int j_y = balls[ball_j].Pos.y / GridSize;
					int j_z = (balls[ball_j].Pos.z + ZRange) / GridSize;
					int home_j = j_x << 16 | j_y << 8 | j_z;

					//�?有这样才�?�?
					if(home_i < home_j)
					{
						if (JudgeCollision(balls[ball_i], balls[ball_j]))
						{
							ChangeSpeed(balls[ball_i], balls[ball_j]);
						}
					}
				}
			}
		}

	}



}

/*
描述：�?�撞检测和处理函数
参数：cell和object数组，ball数组，球和cell的个数，index数组和个数，线程信息，场�?的各种限制和格子信息
返回：无，但�?进�?��?�撞检测和处理
*/
void HandleCollision(uint32_t *cells, uint32_t *objects, Ball* balls, int num_balls, int num_cells,
	uint32_t* indices, uint32_t num_indices, unsigned int num_blocks, unsigned int threads_per_block,
	float XRange, float ZRange, float Height, float GridSize, int GridX, int GridY, int GridZ)
{
	unsigned int threads_total = num_blocks * threads_per_block;
	unsigned int group_per_thread = num_indices / threads_total + 1;
	HandleCollisionCuda <<< num_blocks, threads_per_block >>> (cells, objects, balls, num_balls, num_cells,
		indices, num_indices, group_per_thread,
		XRange, ZRange, Height, GridSize, GridX, GridY, GridZ);
}

/*
描述：空间划分算法�?�理碰撞检测和速度更新（主函数�?
参数：球列表，X范围(-X,X),Z范围(-Z,Z),Y范围(0,Y)，格子大小，X格子�?数，Y格子�?数，Z格子�?数，N�?�?
返回：无，但�?更新球列�?
*/
void HandleCollisionGrid(Ball* balls, float XRange, float ZRange, float Height, 
	float GridSize, int GridX, int GridY, int GridZ, int N,
	unsigned int num_blocks, unsigned int threads_per_block)
{

	//申�?�内�?
	unsigned int cell_size = N * 8 * sizeof(uint32_t);

	uint32_t *cells_gpu;
	uint32_t *cells_gpu_temp;
	uint32_t *objects_gpu;
	uint32_t *objects_gpu_temp;
	uint32_t *indices_gpu;
	uint32_t *indices_num_gpu;
	uint32_t *radix_sums_gpu;

	int num_radices = 1 << RADIX_LENGTH;

	hipMalloc((void **)&cells_gpu, cell_size);
	hipMalloc((void **)&cells_gpu_temp, cell_size);
	hipMalloc((void **)&objects_gpu, cell_size);
	hipMalloc((void **)&objects_gpu_temp, cell_size);
	hipMalloc((void **)&indices_gpu, cell_size);
	hipMalloc((void **)&indices_num_gpu, sizeof(uint32_t));
	hipMalloc((void **)&radix_sums_gpu, num_radices * sizeof(uint32_t));


	
	//初�?�化cell和object
	InitCellKernel <<< num_blocks, threads_per_block, threads_per_block * sizeof(unsigned int) >>> (cells_gpu, objects_gpu, balls, N, XRange, ZRange, Height, GridSize, GridX, GridY, GridZ);

	//基数排序
	SortCells(cells_gpu, objects_gpu, cells_gpu_temp, objects_gpu_temp, radix_sums_gpu, 
		8 * N, indices_gpu, indices_num_gpu, num_blocks, threads_per_block);
	


	uint32_t indices_num;
	hipMemcpy((void*)&indices_num, (void*)indices_num_gpu, sizeof(uint32_t), hipMemcpyDeviceToHost);
	
	HandleCollision(cells_gpu, objects_gpu, balls, N, 8 * N, indices_gpu, indices_num,
		num_blocks, threads_per_block,
		XRange, ZRange, Height, GridSize, GridX, GridY, GridZ);
	

	hipFree(cells_gpu);
	hipFree(cells_gpu_temp);
	hipFree(objects_gpu);
	hipFree(objects_gpu_temp);
	hipFree(indices_gpu);
	hipFree(indices_num_gpu);
	hipFree(radix_sums_gpu);
}


/*
描述：GPU碰撞检�?+运动更新主函数（空间划分算法�?
参数：球列表，一次的时间，X范围(-X,X),Z范围(-Z,Z),Y范围(0,Y)，一�?格子大小，X,Y,Z的格子个数，球个�?
返回：无，但�?更新球列�?
*/
void CollisionDetection(Ball* balls, float TimeOnce, float XRange, float ZRange, float Height, 
	float GridSize, int GridX, int GridY, int GridZ, int N)
{
	//设置，�?�算需要�?�少block和thread
	unsigned int num_blocks = 128;
	unsigned int threads_per_block = 512;
	unsigned int object_size = (N - 1) / threads_per_block + 1;
	if (object_size < num_blocks) {
		num_blocks = object_size;
	}

	Ball* balls_gpu;
	unsigned int nBytes = N * sizeof(Ball);
	hipMalloc((void**)&balls_gpu, nBytes);


	// 初�?�化数据
	hipMemcpy((void*)balls_gpu, (void*)balls, nBytes, hipMemcpyHostToDevice);

	// 执�?�kernel
	HandleCollisionGrid(balls_gpu, XRange, ZRange, Height, GridSize, GridX, GridY, GridZ, N, num_blocks, threads_per_block);
	// 同�??device 保证结果能�?�确访问
	hipDeviceSynchronize();

	// 执�?�kernel
	UpdateBallsMove <<< num_blocks, threads_per_block >>> (balls_gpu, TimeOnce, XRange, ZRange, Height, N);
	// 同�??device 保证结果能�?�确访问
	hipDeviceSynchronize();

	// 记录结果
	hipMemcpy((void*)balls, (void*)balls_gpu, nBytes, hipMemcpyDeviceToHost);

	// 释放内存
	hipFree(balls_gpu);
}